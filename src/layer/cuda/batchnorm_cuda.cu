#include "hip/hip_runtime.h"
//
// Author: Marko Atanasievski
//
// Copyright (C) 2020 TANCOM SOFTWARE SOLUTIONS Ltd. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_util.h"
#include "mat.h"

#include <iostream>

__global__ void gpu_batchnorm_load_model(int channels, float eps, float* a_data_gpu, float* b_data_gpu,
                                         float* bias_data_gpu, float* slope_data_gpu, float* mean_data_gpu, float* var_data_gpu)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= channels) return;
    const float sqrt_var = static_cast<float>(sqrt(var_data_gpu[i] + eps));
    a_data_gpu[i] = bias_data_gpu[i] - slope_data_gpu[i] * mean_data_gpu[i] / sqrt_var;
    b_data_gpu[i] = slope_data_gpu[i] / sqrt_var;
}

// input is 1 dimension
__global__ void gpu_batchnorm_forward_inplace_1(float* d_input, const float* b_data, const float* a_data, const ncnn::CudaMatInfo mat_info, const int input_size)
{

    const int i = blockIdx.x * blockDim.x + threadIdx.x; //limited to 1024 rows
    if (i >= input_size) return;

    d_input[i] =  b_data[i] * d_input[i] + a_data[i];
}

__global__ void gpu_batchnorm_forward_inplace_2(float* d_input, const float* b_data, const float* a_data, const ncnn::CudaMatInfo mat_info, const int input_size)
{
    if (blockIdx.x * blockDim.x + threadIdx.x >= input_size) return;

    const int row = (blockIdx.x * blockDim.x + threadIdx.x) / mat_info.w;
    const int column = (blockIdx.x * blockDim.x + threadIdx.x) % mat_info.w;

    float* ptr = (float*)((unsigned char*)d_input + mat_info.w * mat_info.elemsize * row);

    ptr[column] = b_data[row]*ptr[column]+a_data[row];

}

__global__ void gpu_batchnorm_forward_inplace_3(float* d_input, const float* b_data, const float* a_data, const ncnn::CudaMatInfo mat_info, const int input_size)
{
    if (blockIdx.x * blockDim.x + threadIdx.x >= input_size) return;

    const int channelSize = mat_info.cstep;
    const int channel = (blockIdx.x * blockDim.x + threadIdx.x) / channelSize;
    const int row = ((blockIdx.x * blockDim.x + threadIdx.x) - (channel*channelSize)) / mat_info.w;
    const int column = ((blockIdx.x * blockDim.x + threadIdx.x) - (channel*channelSize)) % mat_info.w;


    const int step = channel * mat_info.cstep * mat_info.elemsize;
    float* ptr = (float*)((unsigned char*)d_input + step);

    const int i = row * mat_info.w+column;
    ptr[i] = b_data[channel] * ptr[i] + a_data[channel];
}

namespace ncnn {


int batchnorm_cuda_load_model(int channels, float eps, float* a_data_gpu, float* b_data_gpu,
                         float* bias_data_gpu, float* slope_data_gpu, float* mean_data_gpu, float* var_data_gpu)
{

    gpu_batchnorm_load_model<<<1, channels>>>(channels, eps, a_data_gpu, b_data_gpu,
                                                bias_data_gpu, slope_data_gpu, mean_data_gpu,
                                                var_data_gpu);

    return 0;
}




int batchnorm_cuda_forward_inplace(float* d_input, const float* b_data, const float* a_data, const CudaMatInfo& matInfo)
{


    if (matInfo.dims == 1)
    {
        const int input_size = matInfo.w;
        int thread_per_block = (((input_size - 1) / 32) + 1) * 32;
        if (thread_per_block > 1024) thread_per_block = 1024;
        dim3 block_size(thread_per_block,1,1);
        dim3 grid_size((matInfo.w - 1) / thread_per_block + 1, 1, 1);
        gpu_batchnorm_forward_inplace_1<<<grid_size, block_size>>>(d_input, b_data, a_data, matInfo, input_size);
    }
    if (matInfo.dims == 2)
    {
        const int input_size = matInfo.w * matInfo.h;
        int thread_per_block = (((input_size - 1) / 32) + 1) * 32;
        if (thread_per_block > 1024) thread_per_block = 1024;
        dim3 block_size(thread_per_block,1,1);
        dim3 grid_size( (input_size - 1) / thread_per_block + 1, 1, 1);
        gpu_batchnorm_forward_inplace_2<<<grid_size, block_size>>>(d_input, b_data, a_data, matInfo, input_size);
    }
    if (matInfo.dims == 3)
    {
        const int total_input_size = matInfo.cstep * matInfo.c;
        int thread_per_block = (((total_input_size - 1) / 32) + 1) * 32;
        if (thread_per_block > 1024) thread_per_block = 1024;
        dim3 block_size(thread_per_block,1,1);
        dim3 grid_size((total_input_size - 1) / thread_per_block + 1, 1, 1);
        gpu_batchnorm_forward_inplace_3<<<grid_size, block_size>>>(d_input, b_data, a_data, matInfo, total_input_size);
    }

    return 0;
}

} // namespace ncnn
