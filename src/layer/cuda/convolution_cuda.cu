#include "hip/hip_runtime.h"
//
// Author: Marko Atanasievski
//
// Copyright (C) 2020 TANCOM SOFTWARE SOLUTIONS Ltd. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.
//



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_util.h"

#include <iostream>

#include "mat.h"

#include "convolution_cuda.h"


static __device__ inline signed char float2int8(float v)
{
    int int32 = static_cast<int>(round(v));
    if (int32 > 127) return 127;
    if (int32 < -127) return -127;
    return (signed char)int32;
}

__global__ void gpu_convolution_cuda_forward(const float* a_input, const ncnn::CudaMatInfo a_info,
                                              const float* weight_data, const ncnn::CudaMatInfo weight_info,
                                              const float* bias_data, const float* activation_params,
                                              float* output, const ncnn::CudaMatInfo output_info,
                                              const ncnn::Convolution_cuda::Convolution_info product_info,
                                              const int* const gpu_space_offset) {

    const int output_column = blockIdx.x * blockDim.x + threadIdx.x;
    const int output_row = blockIdx.y * blockDim.y + threadIdx.y;
    const int num_output = blockIdx.z * blockDim.z + threadIdx.z;


    extern __shared__ float buffer[];
    float* shared_kptr = buffer;

    const int k_index = threadIdx.x;

    if (k_index < product_info.maxk)
    {
        const float* kptr = (const float*)weight_data + product_info.maxk * a_info.c * num_output;
        for (int input_channel = 0; input_channel < a_info.c; input_channel++)
        {
            shared_kptr[input_channel * product_info.maxk + k_index] = kptr[input_channel * product_info.maxk + k_index];
        }
    }

    __syncthreads();

    if (output_column >= output_info.w || output_row >= output_info.h || num_output >= output_info.c)
    {
        return;
    }

    float sum = 0.f;
    if (product_info.bias_term)
    {
        sum += bias_data[num_output];
    }

    for (int input_channel = 0; input_channel < a_info.c; input_channel++)
    {
        const float* sptr = a_input + input_channel * a_info.cstep + output_row * product_info.stride_h * a_info.w + output_column * product_info.stride_w;

        for (int k = 0; k < product_info.maxk; k++)
        {
            const float val = sptr [gpu_space_offset[k]];
            const float w = shared_kptr[input_channel * product_info.maxk + k];
            sum += val * w;
        }
    }

    if (product_info.activation_type == 1)
    {
        sum = max(sum, 0.f);
    }
    else if (product_info.activation_type == 2)
    {
        float slope = activation_params[0];
        sum = sum > 0.f ? sum : sum * slope;
    }
    else if (product_info.activation_type == 3)
    {
        float min = activation_params[0];
        float max = activation_params[1];
        if (sum < min)
            sum = min;
        if (sum > max)
            sum = max;
    }
    else if (product_info.activation_type == 4)
    {
        sum = static_cast<float>(1.f / (1.f + exp(-sum)));
    }
    else if (product_info.activation_type == 5)
    {
        sum = static_cast<float>(sum * tanh(log(exp(sum) + 1.f)));
    }

    const int output_index = num_output * output_info.cstep + output_row * output_info.w + output_column;
    output[output_index] = sum;

}

__global__ void gpu_convolution_cuda_forward_int8(const signed char* a_input, const ncnn::CudaMatInfo a_info,
                                             const float* weight_data, const ncnn::CudaMatInfo weight_info,
                                             const float* bias_data, const float* activation_params,
                                             signed char* output, const ncnn::CudaMatInfo output_info,
                                             const ncnn::Convolution_cuda::Convolution_info product_info,
                                             const int* const gpu_space_offset,
                                             const float *gpu_weight_data_int8_scales) {

    const int output_column = blockIdx.x * blockDim.x + threadIdx.x;
    const int output_row = blockIdx.y * blockDim.y + threadIdx.y;
    const int num_output = blockIdx.z * blockDim.z + threadIdx.z;


    extern __shared__ signed char buffer_int8[];
    signed char* shared_kptr = buffer_int8;

    const int k_index = threadIdx.x;

    if (k_index < product_info.maxk)
    {
        const signed char* kptr = (const signed char*)weight_data + product_info.maxk * a_info.c * num_output;
        for (int input_channel = 0; input_channel < a_info.c; input_channel++)
        {
            shared_kptr[input_channel * product_info.maxk + k_index] = kptr[input_channel * product_info.maxk + k_index];
        }
    }

    __syncthreads();

    if (output_column >= output_info.w || output_row >= output_info.h || num_output >= output_info.c)
    {
        return;
    }

    int sum = 0;

    for (int input_channel = 0; input_channel < a_info.c; input_channel++)
    {
        const signed char* sptr = a_input + input_channel * a_info.cstep + output_row * product_info.stride_h * a_info.w + output_column * product_info.stride_w;

        for (int k = 0; k < product_info.maxk; k++)
        {
            const int val = sptr [gpu_space_offset[k]];
            const int w = shared_kptr[input_channel * product_info.maxk + k];
            sum += val * w;
        }
    }

    const int output_index = num_output * output_info.cstep + output_row * output_info.w + output_column;

    if (product_info.use_int8_requantize)
    {
        // requantize and relu
        float scale_in;
        if (gpu_weight_data_int8_scales[num_output] == 0)
            scale_in = 0;
        else
            scale_in = 1.f / (*product_info.gpu_bottom_blob_int8_scale * gpu_weight_data_int8_scales[num_output]);

        float sumfp32 = sum * scale_in;

        if (product_info.bias_term)
            sumfp32 += bias_data[num_output];

        float scale_out = *product_info.gpu_top_blob_int8_scale;

        signed char sums8 = float2int8(sumfp32 * scale_out);

        if (product_info.activation_type == 1)
        {
            sums8 = max(sums8, (signed char)0);
        }

        output[output_index] = sums8;
    }
    else
    {
        // dequantize and relu
        float scale_in;
        if (gpu_weight_data_int8_scales[num_output] == 0)
            scale_in = 0;
        else
            scale_in = 1.f / (*product_info.gpu_bottom_blob_int8_scale * gpu_weight_data_int8_scales[num_output]);

        float sumfp32 = sum * scale_in;
        if (product_info.bias_term)
            sumfp32 += bias_data[num_output];

        if (product_info.activation_type == 1)
        {
            sumfp32 = max(sumfp32, 0.f);
        }

        ((float*)output)[output_index] = sumfp32;
    }



}


namespace ncnn {

int convolution_cuda_forward(const CudaMat& bottom_blob, CudaMat& top_blob, const Convolution_cuda::Convolution_info& info)
{
    const int number_of_threads = top_blob.w > info.maxk ? top_blob.w : info.maxk;
    int thread_per_block_x = ((number_of_threads - 1) / 32 + 1) * 32;
    if (thread_per_block_x > 64) thread_per_block_x = 64;
    int thread_per_block_y = ((top_blob.h - 1) / 8 + 1) * 8;
    if (thread_per_block_y > 8) thread_per_block_y = 8;
    const int thread_per_block_z = 1;
    const int total_number_of_channels = top_blob.c;
    const int total_number_of_columns = top_blob.w;
    const int total_number_of_rows = top_blob.h;

    const dim3 block_size(thread_per_block_x, thread_per_block_y, thread_per_block_z);
    const dim3 grid_size((total_number_of_columns - 1) / thread_per_block_x + 1,
                         (total_number_of_rows - 1) / thread_per_block_y + 1,
                         (total_number_of_channels - 1) / thread_per_block_z + 1);

    const ncnn::CudaMatInfo bottom_blob_info{bottom_blob};
    const ncnn::CudaMatInfo top_blob_info{top_blob};
    const ncnn::CudaMatInfo weight_info{*info.gpu_weight_data};

    gpu_convolution_cuda_forward<<<grid_size, block_size, bottom_blob.c * info.maxk * sizeof(float)>>>(static_cast<const float*>(bottom_blob.get_craw_data()),
                                                                                    bottom_blob_info,
                                                                                    static_cast<const float*>(info.gpu_weight_data->get_craw_data()),
                                                                                    weight_info,
                                                                                    static_cast<const float*>(info.gpu_bias_data->get_craw_data()),
                                                                                    static_cast<const float*>(info.gpu_activation_params->get_craw_data()),
                                                                                    static_cast<float*>(top_blob.get_raw_data()),
                                                                                    top_blob_info,
                                                                                    info,
                                                                                    static_cast<const int*>(info.gpu_space_ofs));

    return 0;
}

int convolution_cuda_forward_int8(const CudaMat& bottom_blob, CudaMat& top_blob, const Convolution_cuda::Convolution_info& info)
{
    const int number_of_threads = top_blob.w > info.maxk ? top_blob.w : info.maxk;
    int thread_per_block_x = ((number_of_threads - 1) / 64 + 1) * 64;
    if (thread_per_block_x > 128) thread_per_block_x = 128;
    int thread_per_block_y = ((top_blob.h - 1) / 8 + 1) * 8;
    if (thread_per_block_y > 8) thread_per_block_y = 8;
    const int thread_per_block_z = 1;
    const int total_number_of_channels = top_blob.c;
    const int total_number_of_columns = top_blob.w;
    const int total_number_of_rows = top_blob.h;

    const dim3 block_size(thread_per_block_x, thread_per_block_y, thread_per_block_z);
    const dim3 grid_size((total_number_of_columns - 1) / thread_per_block_x + 1,
                         (total_number_of_rows - 1) / thread_per_block_y + 1,
                         (total_number_of_channels - 1) / thread_per_block_z + 1);

    const ncnn::CudaMatInfo bottom_blob_info{bottom_blob};
    const ncnn::CudaMatInfo top_blob_info{top_blob};
    const ncnn::CudaMatInfo weight_info{*info.gpu_weight_data};

    gpu_convolution_cuda_forward_int8<<<grid_size, block_size, bottom_blob.c * info.maxk * sizeof(signed char)>>>(static_cast<const signed char*>(bottom_blob.get_craw_data()),
                                                                                                       bottom_blob_info,
                                                                                                       static_cast<const float*>(info.gpu_weight_data->get_craw_data()),
                                                                                                       weight_info,
                                                                                                       static_cast<const float*>(info.gpu_bias_data->get_craw_data()),
                                                                                                       static_cast<const float*>(info.gpu_activation_params->get_craw_data()),
                                                                                                       static_cast<signed char*>(top_blob.get_raw_data()),
                                                                                                       top_blob_info,
                                                                                                       info,
                                                                                                       static_cast<const int*>(info.gpu_space_ofs),
                                                                                                       static_cast<const float*>(info.gpu_weight_data_int8_scales->get_craw_data()));

    return 0;
}


}
