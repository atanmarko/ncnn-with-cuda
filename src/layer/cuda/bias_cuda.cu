#include "hip/hip_runtime.h"
//
// Author: Marko Atanasievski
//
// Copyright (C) 2020 TANCOM SOFTWARE SOLUTIONS Ltd. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_util.h"
#include "mat.h"

#include <iostream>


__global__ void gpu_bias_forward_inplace(float* a_input, const ncnn::CudaMatInfo a_info, const float* bias) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x; //limited to 1024 rows
    const int input_size = a_info.c * a_info.cstep;
    if (index >= input_size) return;

    const int channel = index / a_info.cstep;
    a_input[index] = a_input[index] + bias[channel];
}

namespace ncnn {

int bias_cuda_forward_inplace(float* a_input, const ncnn::CudaMatInfo& a_info, const float* bias)
{
    int thread_per_block = ((a_info.total_size() / 32) + 1) * 32;
    const dim3 block_size(thread_per_block, 1, 1);
    const dim3 grid_size(a_info.total_size() / thread_per_block + 1, 1, 1);

    gpu_bias_forward_inplace<<<grid_size, block_size>>>(a_input, a_info, bias);

    return 0;
}



}